#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "book.h"
#include "cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ float julia( int x, int y ) {
    const float scale = -2.0;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    //hipComplex c(-0.8, 0.156); // IM01
    //hipComplex c(-0.4, 0.6); // IM02
    //hipComplex c(-1.0, 0.0); // IM03
    //hipComplex c(0.25, 0.0); // IM04
    //hipComplex c(jx, jy); // Mandelbrot
    //hipComplex c(-1.77578, 0.0); // IM06
    //hipComplex c(0.285, 0.0); // IM07
    //hipComplex c(0.285, 0.01); // IM08
    hipComplex c(-0.6, 0.0); // IM09

    hipComplex z(jx, jy);
    //hipComplex z(0.0, 0.0);

    int i = 0;
    for (i=0; i < 2000; i++) {
        z = z * z + c;
        if (z.magnitude2() > 100)
            return i;
    }

    return z.magnitude2();
}

__global__ void kernel( unsigned char *ptr ) {
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    float juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0 * juliaValue;
    ptr[offset*4 + 2] = 128 * juliaValue;
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
    data.dev_bitmap = dev_bitmap;

    dim3    grid(DIM,DIM);
    kernel<<<grid, 1>>>( dev_bitmap );

    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );
                              
    HANDLE_ERROR( hipFree( dev_bitmap ) );
                              
    bitmap.display_and_exit();
}

