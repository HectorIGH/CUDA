#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 100

__global__ void vector_add(float* input) {
	const int tid = threadIdx.x;
	auto step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const auto fst = tid * step_size * 2;
			const auto snd = fst + step_size;
			input[fst] += input[snd];
            __syncthreads();
		}

		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}

int main(){
    float *a;
    float *d_a;
    float result;

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = i;//1.0f;
    }

    // Allocate device memory 
    hipMalloc((void**)&d_a, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);


    // Executing kernel 
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);

    vector_add<<<grid_size, block_size>>>(d_a);
    
    // Transfer data back to host memory
    hipMemcpy(&result, d_a, sizeof(float), hipMemcpyDeviceToHost);

    printf("The sum is: %.2f\n", result);

    // Deallocate device memory
    hipFree(d_a);

    // Deallocate host memory
    free(a); 
}
