#include "hip/hip_runtime.h"
/*
* Compile using following structure
* nvcc -rdc=true -arch compute_35 pi_buffon_needles.cu -o pibn
* and profile with nvprof --unified-memory-profiling off ./pibn
*/
#include <hip/hip_runtime.h>
#include "book.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

// Thread block size
#define THROWS_PER_THREAD 1024
#define BLOCKS 1024
#define THREADS 1024
#define PI 3.1415926535 //known value of PI
#define M_PI 3.14159265358979323846

__global__ void gpu_needles(float *estimate, hiprandState *states) {

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int hits = 0;
    float x, theta;

    hiprand_init(1234 * tid, tid, 0, &states[tid]); // Initialize CURAND

    for(int i = 0; i < THROWS_PER_THREAD; i++) {
        x = abs(hiprand_uniform(&states[tid])) ; // Mapping from [-1, 1] to [0, 1]
        theta = abs(hiprand_uniform(&states[tid])) * 0.5 * M_PI; // Mapping from [-1, 1] to [0, PI / 2]
        hits += (x <= 0.5 * sin(theta)); // Count if the crossing condition is satisfied
    }

    if (hits) {
        estimate[tid] = THROWS_PER_THREAD  / (float) hits; // Threads estimates of PI
    } else {
        estimate[tid] = 0.0;
    }
}

int main( void ) {

    float *host_estimate;
    float *device_estimate;
    hiprandState *devStates;

    // Allocate host memory
    host_estimate = (float*)malloc(sizeof(float) * BLOCKS * THREADS);

    // Allocate device memory 
    HANDLE_ERROR( hipMalloc(&device_estimate, sizeof(float) * BLOCKS * THREADS) );
    HANDLE_ERROR( hipMalloc(&devStates, sizeof(hiprandState) * BLOCKS * THREADS) );

    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // Executing kernel
    gpu_needles<<<BLOCKS, THREADS>>>(device_estimate, devStates);

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );

    // Copying data from device to host
    HANDLE_ERROR( hipMemcpy(host_estimate, device_estimate, sizeof(float) * BLOCKS * THREADS, hipMemcpyDeviceToHost) );

    // Freeing resources
    HANDLE_ERROR( hipFree(device_estimate) );
    HANDLE_ERROR( hipFree(devStates) );

    float PI_BY_GPU = 0;
    for(int i = 0; i < BLOCKS * THREADS; i++) {
        PI_BY_GPU += host_estimate[i];
    }
    PI_BY_GPU /= (BLOCKS * THREADS);

    printf( "\nTime taken:  %3.10f ms to calculate PI as %1.10f with an error of %1.10f.\n", elapsedTime, PI_BY_GPU, (PI_BY_GPU - PI) / PI);
}