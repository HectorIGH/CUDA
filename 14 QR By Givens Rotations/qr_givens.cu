#include "hip/hip_runtime.h"
/*
* Compile using following structure
* nvcc -rdc=true -arch compute_35 qr_givens.cu -o gr
* and profile with nvprof --unified-memory-profiling off ./gr
*/
#include <hip/hip_runtime.h>
#include "book.h"
#include <hip/hip_runtime.h>

extern const int N = 8;		//this defines the number of elements in each vector
extern const int M = 8;		//this defines the number of vectors that need to be orthogonalized

void host_traspose(double *a) {
    double aux[N * N];
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            aux[i + N * j] = a[i * N + j];
        }
    }

    for(int i = 0; i < N * N; i++) {
        a[i] = aux[i];
    }
}

__global__ void solver(double *a, double *b, double *x, double *hold, int bid) {

    int tid = threadIdx.x;

    int index = N - bid;

    int last = (N * N - 1);

    __syncthreads();
    //hold[tid] = x[index + tid] * a[last - N * bid - tid];
    hold[tid] = x[N - 1 - tid] * a[last - N * bid - tid];

    //printf("\nProduct of a with: %f x %f in index %d for thread %d results in %f\n", a[last - N * bid - tid], x[N - 1 - tid], N - 1 - tid, tid, hold[tid]);


    //printf("\nCoeficient: %f", hold[index]);
    if(tid == 0) {
        double sum = 0;
        for (int i = 0; i < bid; i++) {
            sum += hold[i];
        }
        //printf("\nSum is %f and b %f and substract %f\n", sum, b[N - 1 - bid], b[N - 1 - bid ] - sum);
        x[N - 1 - bid] = (b[index + tid - 1] - sum) / a[last - N * bid - bid];
        b[N - 1 - bid] = x[N - 1 - bid];
        //printf("\nFinally coeficient: %f", x[N - 1 - bid]);
    }
    __syncthreads();
}

__global__ void product_v(double *a, double *b, double *c) {

    int row = threadIdx.x;

    float element = 0;
    for(int i = 0; i < N; ++i) {
        element += a[row * N + i] * b[i];
    }
    c[row] = element;
}

__global__ void diagonal_inverse(double *matrix) {
    int tid = threadIdx.x;
    int index = N * tid + tid;
    matrix[index] = 1 / matrix[index];
}

__global__ void traspose(double *a) {
    double aux[N * N];
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            aux[i + N * j] = a[i * N + j];
        }
    }

    for(int i = 0; i < N * N; i++) {
        a[i] = aux[i];
    }
}


__global__ void product(double *a, double *b, double *c) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float element = 0;
    for(int i = 0; i < N; ++i) {
        element += a[row * N + i] * b[i * N + col];
    }
    c[row * N + col] = element;
}


__global__ void givens(double *A, double *Qt, double *R, double *G, int col) {
    int row = threadIdx.x + 1 + col;
    if(col < row) { // Below diagonal
        __syncthreads();
        //printf("From thread %d and col %d Zeroing element %f with y = %f and x = %f \n", row, col, R[N * row + col], R[N * row + col], R[N * col + col]);

        
        
        double theta = atan(-(R[N * row + col]) / (R[N * col + col]));
        //printf("Cos(theta) %f\n", cos(theta));
        G[col * N + col] = cos(theta);
        G[row * N + row] = cos(theta);
        G[N * row + col] = sin(theta);
        G[col * N + row] = -sin(theta);

        
        dim3 dimBlockG(N, N);
        dim3 dimGridG(1, 1);
        product<<<dimGridG, dimBlockG>>>(G, R, R);
        
        product<<<dimGridG, dimBlockG>>>(Qt, G, Qt);
        __syncthreads();
    }
}


int main( void ) {
    // Input in row order
    //double input[N * N] = {12, -51, 4, 6, 167, -68, -4, 24, -41};
    //double a[N * N] = {12, -51, 4, 6, 167, -68, -4, 24, -41};
    //double r[N * N] = {12, -51, 4, 6, 167, -68, -4, 24, -41};
    //double b[N] = {2, 3, 5};

    //double input[N * N] = {4, 4, 2, 4, 5, 3, 2, 3, 3};
    //double a[N * N] = {4, 4, 2, 4, 5, 3, 2, 3, 3};
    //double r[N * N] = {4, 4, 2, 4, 5, 3, 2, 3, 3};
    //double b[N] = {2, 3, 5};

    
    double input[N * N];
    double a[N * N];
    double r[N * N];
    double b[N];
    

    //srand((unsigned) time(NULL));
    
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            input[i * N + j] = rand() % 10 + 1;
            a[i * N + j] = input[i * N + j];
            r[i * N + j] = input[i * N + j];
        }
        b[i] = rand() % 10 + 1;
    }


    double *host_d;
    double *host_y;
    double *host_x;

    host_d = (double*)malloc(sizeof(double) * M * N);
    host_y = (double*)malloc(sizeof(double) * N);
    host_x = (double*)malloc(sizeof(double) * N);

    double host_eye[N * N];
    double host_G[N * N];

    for(int i = 0; i < N * N; i++) {
        host_eye[i] = 0;
        host_G[i] = 0;
    }
    for(int i = 0; i < N; i++) {
        host_eye[N * i + i] = 1;
        host_G[N * i + i] = 1;
    }

    double *dev_R;
    double *dev_Q_t;
    double *dev_D;
    double *dev_A;
	double *dev_input;
	double *dev_m;
    double *dev_y;
    double *dev_aux;
    double *dev_b;
    double *dev_x;
    double *G;

	HANDLE_ERROR(hipMalloc((void**)&dev_input, (M*N)*sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_m, sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_R, (M*N)*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_Q_t, (M*N)*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_D, (M*N)*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_A, (M*N)*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_aux, (M*N)*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_y, N * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_x, N * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&G, (N * N) * sizeof(double)));

    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(dev_input, input, (M * N) * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_R, input, (M * N) * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_Q_t, host_eye, (N * N) * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(G, host_G, (N * N) * sizeof(double), hipMemcpyHostToDevice));

    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

	for(int col = 0; col < N - 1; col++){
		givens<<<1, N - 1 - col>>>(dev_input, dev_Q_t, dev_R, G, col);
	}

	
    // Solve linear equations
	
	HANDLE_ERROR(hipMemcpy(input, dev_Q_t, M*N*sizeof(double), hipMemcpyDeviceToHost));
    // Here input and dev_input holds the transpose of Q. Q^T

    HANDLE_ERROR( hipMemcpy(r, dev_R, sizeof(double) * M * N, hipMemcpyDeviceToHost) );

    HANDLE_ERROR( hipMemcpy(dev_input, dev_Q_t, sizeof(double) * M * N, hipMemcpyDeviceToDevice));
    
    // Calculating Q_t * Q = D
    //traspose<<<1, 1>>>(dev_Q_t); // Not necessary since dev_input and hence dev_Q_t already holds the transpose.
    traspose<<<1, 1>>>(dev_input); // We get the real Q matrix
    dim3 dimBlock(N, N);
    dim3 dimGrid(1, 1);
    product<<<dimGrid, dimBlock>>>(dev_Q_t, dev_input, dev_D);

    HANDLE_ERROR( hipMemcpy(host_d, dev_D, sizeof(double) * M * N, hipMemcpyDeviceToHost) );

    // Getting the inverse of D

    diagonal_inverse<<<1, N>>>(dev_D);
    // Finding Y
    product<<<dimGrid, dimBlock>>>(dev_D, dev_Q_t, dev_aux);
    product_v<<<1,N>>>(dev_aux, dev_b, dev_y);

    HANDLE_ERROR( hipMemcpy(host_y, dev_y, sizeof(double) * N, hipMemcpyDeviceToHost) );

    HANDLE_ERROR(hipMemcpy(host_x, dev_y, N * sizeof(double), hipMemcpyDeviceToHost));

    // Finally solving for x. First we need to include the posibility that X_N is not 1

    host_x[N - 1] = host_x[N - 1] / r[N * N - 1]; // To include the possible sing in the reduced matrix
    
    HANDLE_ERROR(hipMemcpy(dev_x, host_x, N * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_y, dev_x, N * sizeof(double), hipMemcpyDeviceToDevice));

    for(int i = 1; i < N; i++) {
        double *dev_hold;
        HANDLE_ERROR(hipMalloc((void**)&dev_hold, (i) * sizeof(double)));
        solver<<<1, i>>>(dev_R, dev_x, dev_y, dev_hold, i);
    }

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
    printf( "\nTime taken:  %3.10f ms\n", elapsedTime );

    HANDLE_ERROR(hipMemcpy(host_x, dev_x, N * sizeof(double), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_input));
	HANDLE_ERROR(hipFree(dev_m));
    HANDLE_ERROR(hipFree(dev_R));
    HANDLE_ERROR(hipFree(dev_A));

    printf("\nOriginal Extended Matrix:\n");
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < M; j++) {
            printf("%f ", a[i * N + j]);
        }
        printf("\t %f\n", b[i]);
    }

    //printf("\nQ Matrix:\n");
    //for(int i = 0; i < N; i++) {
        //for(int j = 0; j < M; j++) {
            //printf("%f ", input[i * N + j]);
        //}
        //printf("\n");
    //}

    //printf("\nR Matrix:\n");
    //for(int i = 0; i < N; i++) {
        //for(int j = 0; j < M; j++) {
            //printf("%f ", r[i * N + j]);
        //}
        //printf("\n");
    //}

    //printf("\nD Matrix:\n");
    //for(int i = 0; i < N; i++) {
        //for(int j = 0; j < M; j++) {
            //printf("%f ", host_d[i * N + j]);
        //}
        //printf("\n");
    //}

    //printf("\nY Vector:\n");
    //for(int i = 0; i < N; i++) {
        //printf("%f ", host_y[i]);
    //}
    //printf("\n");

    //printf("\nX Solutions:\n");
    //for(int i = 0; i < N; i++) {
        //printf("%f ", host_x[i]);
    //}
    //printf("\n");
}