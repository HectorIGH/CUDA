#include "hip/hip_runtime.h"
/*
* Compile using following structure
* nvcc -rdc=true -arch compute_35 binary.cu -o bn
* and profile with nvprof --unified-memory-profiling off ./bn
* check memory error with cuda-memcheck ./bn
*/
#include <hip/hip_runtime.h>
#include "book.h"
#include <hip/hip_runtime.h>


__global__ void binarize(double *N, double *binary) {
    int tid = threadIdx.x;

    binary[tid] = 1 << tid & int(N[0]) ? 1 : 0;

}

__global__ void residuals_coordinates(double *N, double *x, double *residuals) {
    int tid = threadIdx.x;

    residuals[tid] = int(powf(x[0], 1 << tid)) % int(N[0]) ;

    //printf("In residuals %f\n\n", powf(x[0], 1 << tid));

}

__global__ void masking(double *binary, double *residuals) {
    int tid = threadIdx.x;

    if(binary[tid] == 0.0) {
        __syncthreads();
        residuals[tid] = 1;
    }
}

__global__ void residual(double *residuals, double *m) {
    int tid = threadIdx.x;
    int number_of_threads = blockDim.x;
    int step_size = 1;

    while(number_of_threads > 0) {
        if(tid < number_of_threads) {
            
            int first = tid * 2 * step_size;
            int second = first + step_size;

            residuals[first] *= residuals[second];
            __syncthreads();
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
    residuals[0] = int(residuals[0]) % int(m[0]);
}


int main( int argc, char *argv[] ) {
    
    double host_x[1] = {6};
    double host_e[1] = {15};
    double host_m[1] = {21};

    int bits = int(log2(host_e[0])) + 1;

    int power_bits = 1;
    while (power_bits < bits){
        power_bits <<= 1;
    }
    bits = power_bits;
    
    double *host_binary;
    double *host_residuals;
    double *host_products;
    double *host_residuo;

    host_binary = (double*)malloc(sizeof(double) * bits);
    host_residuals = (double*)malloc(sizeof(double) * bits);
    host_products = (double*)malloc(sizeof(double) * bits);
    host_residuo = (double*)malloc(sizeof(double) * 1);

    //srand((unsigned) time(NULL));


    double *dev_x;
    double *dev_e;
    double *dev_m;
    
    double *dev_binary;
	double *dev_residuals;
    double *dev_products;

    HANDLE_ERROR(hipMalloc((void**)&dev_x, 1 * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_e, 1 * sizeof(double)));
    HANDLE_ERROR(hipMalloc((void**)&dev_m, 1 * sizeof(double)));
    
    HANDLE_ERROR(hipMalloc((void**)&dev_binary, bits * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_residuals, bits * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_products, bits * sizeof(double)));

    HANDLE_ERROR(hipMemcpy(dev_x, host_x, 1 * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_e, host_e, 1 * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_m, host_m, 1 * sizeof(double), hipMemcpyHostToDevice));



    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    
    binarize<<<1, bits>>>(dev_e, dev_binary);

    residuals_coordinates<<<1, bits>>>(dev_m, dev_x, dev_residuals);

    HANDLE_ERROR(hipMemcpy(dev_products, dev_residuals, bits * sizeof(double), hipMemcpyDeviceToDevice));

    masking<<<1, bits>>>(dev_binary, dev_products);

    residual<<<1, bits / 2>>>(dev_products, dev_m);

	
    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
    printf( "\nTime taken:  %3.10f ms\n", elapsedTime );


    HANDLE_ERROR(hipMemcpy(host_binary, dev_binary, bits * sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(host_residuals, dev_residuals, bits * sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(host_products, dev_products, bits * sizeof(double), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(host_residuo, dev_products, 1 * sizeof(double), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_x));
    HANDLE_ERROR(hipFree(dev_e));
    HANDLE_ERROR(hipFree(dev_m));
    HANDLE_ERROR(hipFree(dev_binary));
    HANDLE_ERROR(hipFree(dev_residuals));
    HANDLE_ERROR(hipFree(dev_products));

    printf("\nBinary Coordinates Representation of %1.0f:\n\n", host_e[0]);
    for(int i = 0; i < bits; i++) {
        printf("%1.0f ", host_binary[i]);
    }

    printf("\n\nResiduals Coordinates:\n\n");
    for(int i = 0; i < bits; i++) {
        printf("%f ", host_residuals[i]);
    }

    printf("\nProducts:\n");
    for(int i = 0; i < bits; i++) {
        printf("%f ", host_products[i]);
    }

    printf("\n\n[%1.0f ^ %1.0f]_%1.0f = %f\n", host_x[0],host_e[0], host_m[0], host_residuo[0]);
}