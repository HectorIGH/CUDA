#include "hip/hip_runtime.h"
/*
* Compile using following structure
* nvcc -rdc=true -arch compute_35 pi_monte_carlo.cu -o pimc
* and profile with nvprof --unified-memory-profiling off ./pimc
*/
#include <hip/hip_runtime.h>
#include "book.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

// Thread block size
#define TRIALS_PER_THREAD 512
#define BLOCKS 1024
#define THREADS 1024
#define PI 3.1415926535 //known value of PI

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int point_in_circle = 0;
    float x, y;

    hiprand_init(1234, tid, 0, &states[tid]); // Initialize CURAND

    for(int i = 0; i < TRIALS_PER_THREAD; i++) {
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        point_in_circle += (x*x + y*y < 1.0f); // Count if x & y are in the circle
    }
    estimate[tid] = 4.0f * point_in_circle / (float) TRIALS_PER_THREAD; // Threads estimates of PI
}

int main( void ) {

    float *host_estimate;
    float *device_estimate;
    hiprandState *devStates;

    // Allocate host memory
    host_estimate = (float*)malloc(sizeof(float) * BLOCKS * THREADS);

    // Allocate device memory 
    HANDLE_ERROR( hipMalloc(&device_estimate, sizeof(float) * BLOCKS * THREADS) );
    HANDLE_ERROR( hipMalloc(&devStates, sizeof(hiprandState) * BLOCKS * THREADS) );

    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // Executing kernel
    gpu_monte_carlo<<<BLOCKS, THREADS>>>(device_estimate, devStates);

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );

    // Copying data from device to host
    HANDLE_ERROR( hipMemcpy(host_estimate, device_estimate, sizeof(float) * BLOCKS * THREADS, hipMemcpyDeviceToHost) );

    // Freeing resources
    HANDLE_ERROR( hipFree(device_estimate) );
    HANDLE_ERROR( hipFree(devStates) );

    float PI_BY_GPU = 0;
    for(int i = 0; i < BLOCKS * THREADS; i++) {
        PI_BY_GPU += host_estimate[i];
    }
    PI_BY_GPU /= (BLOCKS * THREADS);

    printf( "\nTime taken:  %3.10f ms to calculate PI as %1.10f with an error of %1.10f.\n", elapsedTime, PI_BY_GPU, (PI_BY_GPU - PI) / PI);
}