#include "hip/hip_runtime.h"
/*
* Compile using following structure
* nvcc -rdc=true -arch compute_35 matrix_determinant_minors.cu -o min
* and profile with nvprof --unified-memory-profiling off ./min
*/
#include <hip/hip_runtime.h>
#include "book.h"
#include <hip/hip_runtime.h>

// Thread block size
#define N 3

typedef struct {
    int width;
    int height;
    double* elements;
} Matrix;

__device__ int plop = 0;

__global__ void determinant_by_minors(Matrix matrix, double *determinant, int n, int bloques, int depth, int from) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int new_dimension = n - 1;
    int factor = 1;
    int j = 0;

    printf("Block %d of %d with %d threads from %d and thread %d and k %d\n\n", blockIdx.x, bloques, blockDim.x, from, threadIdx.x, k);

    Matrix aux_matrix;
    aux_matrix.width = new_dimension;
    aux_matrix.height = new_dimension;

    hipMalloc(&aux_matrix.elements, sizeof(double) * aux_matrix.width * aux_matrix.height);

    // Calculating minors
    for(int i = n; i < n * n; i++) {
        if (i == index + n * factor) {
            factor++;
            continue;
        } else {
            aux_matrix.elements[j] = matrix.elements[i];
            j++;
        }
    }

    determinant[k] = aux_matrix.elements[0];
    __syncthreads();
    determinant_by_minors<<<n - 1, n - 2>>>(aux_matrix, determinant, n - 1, n - 1, 0, blockIdx.x + threadIdx.x);
}

int main( void ) {

    Matrix host_matrix;
    host_matrix.width = N;
    host_matrix.height = N;
    double *host_determinant;

    Matrix device_matrix;
    device_matrix.width = host_matrix.width;
    device_matrix.height = host_matrix.height;
    double *device_determinant;

    int factorial = 1;
    for(int i = 1; i <= N; i++) {
        factorial *= i;
    }


    // Allocate host memory
    host_matrix.elements = (double*)malloc(sizeof(double) * host_matrix.width * host_matrix.height);
    host_determinant = (double*)malloc(sizeof(double) * factorial * N);

    for(int i = 0;i < factorial * N; i++) {
        host_determinant[i] = 1;
    }

    // Initialize host Matrix
    for(int i = 0; i < host_matrix.width * host_matrix.height; i++){
        host_matrix.elements[i] = rand() % 100 + 1;
    }

    printf("\nOriginal Matrix:\n");
    for(int i = 0; i < host_matrix.height; i++){
        printf("| ");
        for(int j = 0; j < host_matrix.width; j++) {
            printf("%.0f | ", host_matrix.elements[j + i * host_matrix.width]);
        }
        printf("\n");
    }


    // Allocate device memory
    HANDLE_ERROR( hipMalloc(&device_matrix.elements, sizeof(double) * device_matrix.width * device_matrix.height) );
    HANDLE_ERROR( hipMalloc(&device_determinant, sizeof(double) * factorial * N) );

    // Transfer data from host to device memory
    HANDLE_ERROR( hipMemcpy(device_determinant, host_determinant, sizeof(double)  * factorial * N, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(device_matrix.elements, host_matrix.elements, sizeof(double) * host_matrix.width * host_matrix.height, hipMemcpyHostToDevice) );

    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // Executing kernel

    determinant_by_minors<<<N, N - 1>>>(device_matrix, device_determinant, N, N, 0, 0);

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
    printf( "\nTime taken:  %3.10f ms\n", elapsedTime );

    // Transfer data from device to host memory
    HANDLE_ERROR( hipMemcpy(host_determinant, device_determinant, sizeof(double) * factorial * N, hipMemcpyDeviceToHost) );

    // Free resources
    HANDLE_ERROR( hipFree(device_matrix.elements) );
    HANDLE_ERROR( hipFree(device_determinant) );

    
    for(int i = 0; i < factorial * N; i++) {
        printf("\n Determinant: %1.0f\n", host_determinant[i]);
    }
}